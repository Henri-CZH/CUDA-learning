#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "common/tester.h"

#define LDST64BITS(value) (reinterpret_cast<float2*>(&(value))[0])
#define LDST32BITS(value) (reinterpret_cast<__half2*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])

// only 8 warp per block process matrix mat (64,16)*(16,32)
template<int WMMA_M = 16, int WMMA_N = 16, int WMMA_K = 16, 
        int WMMA_TILE_M = 4, int WMMA_TILE_N = 2, 
        int WARP_TILE_M = 2, int WARP_TILE_N = 4>
__global__ void hgemm_wmma_m16n16k16_mma4x2_warp2x4_kernel(__half *A, __half *B, __half *C,
                                                  int M, int N, int K) {
    // define iternation times in K axis
    const int NUM_K_TIMES = div_ceil(K, WMMA_K);

    // define bid in M, N axis
    const int bid_M = blockIdx.y;
    const int bid_N = blockIdx.x;

    // define tid in M, N axis
    const int tid = threadIdx.x + blockDim.x * threadIdx.y; // 0~255

    // define define warp id in M, N axis->4 warp in M axis, 2 warp in N warp within a block
    const int warp_id = tid / 32; // 0~7
    const int lane_id = tid % 32; // 0~31

    const int warp_m = warp_id / 2; // 0~3
    const int warp_n = warp_id % 2; // 0~1

    // define size of fragment C->(BM, BN)
    constexpr int BM = WMMA_M * WMMA_TILE_M * WARP_TILE_M; // 16x4x2=128
    constexpr int BN = WMMA_N * WMMA_TILE_N * WARP_TILE_N; // 16x2x4=128
    constexpr int BK = WMMA_K; // 16

    // define shared memory w/. [BM][BN]
    __shared__ __half s_a[BM][BK], s_b[BK][BN]; // 128x16x2=4k, 16x128x2=4k

    // define shared memory offset in M, K, N axis
    // 256 thread load s_a = (128,16), s_b = (16,128)
    // s_a, 8 half/thread->2 threads/row, 128 rows->256 threads
    // s_b, 8 half/thread->16 threads/row, 16 rows->256 threads
    const int load_smem_a_m = tid / 2; // 0~127
    const int load_smem_a_k = (tid % 2) * 8; // 0, 8

    const int load_smem_b_k = tid / 16; // 0~15
    const int load_smem_b_n = (tid % 16) * 8; // 0, 16, 32, 48, ..., 120

    // define global offset in M, N axis
    const int load_gmem_a_m = BM * bid_M + load_smem_a_m; // global m
    const int load_gmem_b_n = BN * bid_N + load_smem_b_n; // global n

    if (load_gmem_a_m >= M && load_gmem_b_n >= N) {
        return;
    }

    // define fragment C and initialize
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, __half> C_frag[WARP_TILE_M][WARP_TILE_N];
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
            nvcuda::wmma::fill_fragment(C_frag[i][j], 0.0);
        }
    }
    
    // process
#pragma unroll
    for (int k_id = 0; k_id < NUM_K_TIMES; ++k_id) {
        // define global col offset of matrix A
        int load_gmem_a_k = k_id * WMMA_K + load_smem_a_k;

        // define global row offset of matrix B
        int load_gmem_b_k = k_id * WMMA_K + load_smem_b_k;

        // load data from global memory in shared memory
        LDST128BITS(s_a[load_smem_a_m][load_smem_a_k]) = LDST128BITS(A[load_gmem_a_m * K + load_gmem_a_k]);

        LDST128BITS(s_b[load_smem_b_k][load_smem_b_n]) = LDST128BITS(B[load_gmem_b_k * N + load_gmem_b_n]);

        __syncthreads();

        // define fragment A, B
        nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, nvcuda::wmma::row_major> A_frag[WARP_TILE_M];
        nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, nvcuda::wmma::row_major> B_frag[WARP_TILE_N];

        // load 2 tile->reg, smem a->frags b
#pragma unroll
        for (int i = 0; i < WARP_TILE_M; ++i) {
            const int warp_smem_a_m = warp_m * WMMA_M * WARP_TILE_M + i * WMMA_M;
            nvcuda::wmma::load_matrix_sync(A_frag[i], &s_a[warp_smem_a_m][0], BK); // (WMMA_M,BK)
        }

        // load 4 tile->reg, smem b->frags b
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
            const int warp_smem_b_n = warp_n * WMMA_N * WARP_TILE_N + j * WMMA_N;
            nvcuda::wmma::load_matrix_sync(B_frag[j], &s_b[0][warp_smem_b_n], BN); // (BK,WMMA_N)
        }

#pragma unroll
        for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
            for (int j = 0; j < WARP_TILE_N; ++j) {
                // mma: 2 mma in m axis, 4 mma in n axis
                nvcuda::wmma::mma_sync(C_frag[i][j], A_frag[i], B_frag[j], C_frag[i][j]);
            }
        }        

        __syncthreads();
    }

    // store result in matrix C
#pragma unroll
    for (int i = 0; i < WARP_TILE_M; ++i) {
#pragma unroll
        for (int j = 0; j < WARP_TILE_N; ++j) {
            // store: 2 matrix (16,16) in m axis, 4 matrix (16,16) in n axis
            const int store_gmem_c_m = bid_M * BM + warp_m * WMMA_M * WARP_TILE_M + i * WMMA_M;
            const int store_gmem_c_n = bid_N * BN + warp_n * WMMA_N * WARP_TILE_N + j * WMMA_N;
            nvcuda::wmma::store_matrix_sync(C + store_gmem_c_m * N + store_gmem_c_n, C_frag[i][j], N, nvcuda::wmma::mem_row_major); // (WMMA_M,WMMA_N)
        }
    }

}

void hgemm_wmma_m16n16k16_mma4x2_warp2x4(__half *A, __half *B, __half *C, int M, int N, int K) {
    constexpr int WMMA_M = 16;
    constexpr int WMMA_N = 16;
    constexpr int WMMA_K = 16;
    constexpr int WMMA_TILE_M = 4;
    constexpr int WMMA_TILE_N = 2;
    constexpr int WARP_TILE_M = 2;
    constexpr int WARP_TILE_N = 4;
    dim3 block(256);
    dim3 grid(div_ceil(N, WMMA_N * WMMA_TILE_N * WARP_TILE_N), div_ceil(M, WMMA_M * WMMA_TILE_M * WARP_TILE_M));

    hgemm_wmma_m16n16k16_mma4x2_warp2x4_kernel<WMMA_M, WMMA_N, WMMA_K, WMMA_TILE_M, WMMA_TILE_N><<<grid, block>>>(A, B, C, M, N, K);
}


int main(int argc, char *argv[]) {
    Tester tester(512, 2048, 1024, 1, 10, 100, true);
    tester.evaluate(hgemm_wmma_m16n16k16_mma4x2_warp2x4, "my_hgemm_v1_wmma_m16n16k16_mma4x2_warp2x4");
}