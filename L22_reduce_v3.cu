#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>

// v3: one tid operate 4 data per time and reduce half of number of block

template<int blockSize> // blockSize as template arg use for static shared memory size apply during compile phase
__global__ void reduce_v3(const int* d_in, int* d_out, const int n)
{   
    int tid = threadIdx.x;
    int gtid = threadIdx.x + blockIdx.x * (blockSize * 2); // global thread idx
    __shared__ float smem[blockSize]; // declare shared memory 
    smem[tid] = d_in[gtid] + d_in[gtid + blockSize]; // load two near block data into shared memory corresponding to thread gtid; smem[0]<-d_in[0] + d_in[256]
    __syncthreads(); // synchronize all threads in a block

    for(int idx = blockDim.x / 2; idx > 0; idx >>=1)
    {
        // method 1:
        // here is no warp divergent, because no use threads is idle
        // if(tid % (2 * idx) == 0)
        // if(tid & (2 * idx - 1) == 0)
        //     smem[tid]+= d_in[tid + idx];

        // method 2: 
        // here is bank conflict, because there is 32 bank memory, tid0 operate sdata[0]:bank0 and sdata[1]:bank1, tid16 operate sdata[32]:bank0 and sdata[33]:bank1;
        // unsigned s = 2 * idx * tid;
        // if(s < blockDim.x)
        //     smem[s] += smem[s + idx];
        if(tid < idx)
            smem[tid] += smem[tid + idx]; // tid0 operate smem[0]<-smem[0] + smem[128], smem[0]<-d_in[0] + d_in[256] + d_in[128] + d_in[384] all tid > 128 in a block are idle
        
        __syncthreads();
    }

    // all (N / blockSize) block's threads have finished
    if(tid == 0)
        d_out[blockIdx.x] = smem[tid]; // sum of all threads in a block
}

bool checkResult(int* out, int groudtruth, int n)
{   
    float res = 0;
    for(int i = 0; i < n; i++)
        res += out[i];

    if(*out != groudtruth)
        return false;
    
    return true;
}

int main()
{
    float milliseconds = 0;
    constexpr int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 256; // number of threads in a block
    int gridSize = std::min((N + blockSize - 1) / blockSize, deviceProp.maxGridSize[0]); // N = 255->gridSize = 1; N = 257->gridSize = 2

    // allocate CPU and GPU memory
    int *h_in = (int*)malloc(N * sizeof(int));
    int *d_in;
    hipMalloc((void**)&d_in, N * sizeof(int));

    int *h_out = (int*)malloc(gridSize * sizeof(int));
    int *d_out;
    hipMalloc((void**)&d_out, gridSize * sizeof(int));

    // initialize data
    for(int i = 0; i < N; i++)
    {
        h_in[i] = 1;
    }

    int groundTruth = N * 1;

    hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);

    // allocate block and thread size
    dim3 grid(gridSize);
    dim3 block(blockSize); // thrad size

    // record GPU execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // allocate 1 block, 1 thread
    reduce_v3<blockSize / 2><<<grid, block>>>(d_in, d_out, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("allocated %d blocks, data counts are %d\n", gridSize, N);

    bool is_right = checkResult(h_out, groundTruth, gridSize);
    if(is_right)
    {
        printf("the ans is right\n");
    }
    else
    {
        printf("the ans is wrong\n");
        // for(int i = 0; i < gridSize; i++)
        // {
        //     printf("res per block: %1f\n", h_out[i]);
        // }
        printf("groundTruth is %f \n", groundTruth);
    }

    printf("reduce_baseline latency = %f ms\n", milliseconds);

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;

}