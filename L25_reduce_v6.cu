#include <hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>

// v6 one tid operate multiple data per time

__device__ void warpSharedMemReduce(volatile float* smem, int tid)
{
    int x = smem[tid];
    if(blockDim.x >= 64)
    {
        x += smem[tid + 32];
        __syncwarp(); // synchronize all threads in a warp
        smem[tid] = x;
        __syncwarp();
    }

        x += smem[tid + 16];
        __syncwarp();
        smem[tid] = x;
        __syncwarp();
        x += smem[tid + 8];
        __syncwarp();
        smem[tid] = x;
        __syncwarp();
        x += smem[tid + 4];
        __syncwarp();
        smem[tid] = x;
        __syncwarp();
        x += smem[tid + 2];
        __syncwarp();
        smem[tid] = x;
        __syncwarp();
        x += smem[tid + 1];
        __syncwarp();
}

template<int blockSize> // blockSize as template arg use for static shared memory size apply during compile phase
__global__ void reduce_v6(const int* d_in, int* d_out, size_t n)
{   
    int tid = threadIdx.x;
    int gtid = threadIdx.x + blockIdx.x * blockDim.x; // global thread idx, here can use blockSize or blockDim.x 
    __shared__ float smem[blockSize]; // declare shared memory, here can't use blockDim.x, because shared memory size apply is in compile phase
    unsigned int total_thread_num = gridDim.x * blockDim.x;
    int sum = 0;
    for(int i = gtid; i < n; i += total_thread_num)
    {
        if(i > n)
            break;
        sum += d_in[i];
    }

    smem[tid] = sum; // load multiple data into shared memory
    __syncthreads(); // synchronize all threads in a block

    // for(int idx = blockDim.x / 2; idx > 32; idx >>=1)
    // {
    //     // method 1:
    //     // here is no warp divergent, because no use threads is idle
    //     // if(tid % (2 * idx) == 0)
    //     // if(tid & (2 * idx - 1) == 0)
    //     //     smem[tid]+= d_in[tid + idx];

    //     // method 2: 
    //     // here is bank conflict, because there is 32 bank memory, tid0 operate sdata[0]:bank0 and sdata[1]:bank1, tid16 operate sdata[32]:bank0 and sdata[33]:bank1;
    //     // unsigned s = 2 * idx * tid;
    //     // if(s < blockDim.x)
    //     //     smem[s] += smem[s + idx];
    //     if(tid > idx)
    //         smem[tid] += smem[tid + idx]; // tid0 operate smem[0]<-smem[0] + smem[128], smem[0]<-d_in[0] + d_in[256] + d_in[128] + d_in[384] all tid > 128 in a block are idle
        
    //     __syncthreads();
    // }

    // loop unrolling
    if(blockSize >= 1024)
    {
        if(tid < 512)
        {
            smem[tid] += smem[tid + 512];
        }
        __syncthreads();
    }

    if(blockSize >= 512)
    {
        if(tid < 256)
        {
            smem[tid] += smem[tid + 256];
        }
        __syncthreads();
    }

    if(blockSize >= 256)
    {
        if(tid < 128)
        {
            smem[tid] += smem[tid + 128];
        }
        __syncthreads();
    }

    if(blockSize >= 128)
    {
        if(tid < 64)
        {
            smem[tid] += smem[tid + 64];
        }
        __syncthreads();
    }


    // last warp in a block independently calc reduce 
    if(tid < 32)
        warpSharedMemReduce(smem, tid);
    
    // all (N / blockSize) block's threads have finished
    if(tid == 0)
        d_out[blockIdx.x] = smem[tid]; // smem[0]: sum of all threads in a block
}

bool checkResult(int* out, int groudtruth, int n)
{   
    float res = 0;
    for(int i = 0; i < n; i++)
        res += out[i];

    if(*out != groudtruth)
        return false;
    
    return true;
}

int main()
{
    float milliseconds = 0;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int blockSize = 256; // number of threads in a block
    int gridSize = std::min((N + blockSize - 1) / blockSize, deviceProp.maxGridSize[0]); // N = 255->gridSize = 1; N = 257->gridSize = 2

    // allocate CPU and GPU memory
    int *h_in = (int*)malloc(N * sizeof(int));
    int *d_in;
    hipMalloc((void**)&d_in, N * sizeof(int));

    int *h_out = (int*)malloc(gridSize * sizeof(int));
    int *d_out, *part_out;
    hipMalloc((void**)&d_out, 1 * sizeof(int));
    hipMalloc((void**)&part_out, gridSize * sizeof(int));

    // initialize data
    for(int i = 0; i < N; i++)
    {
        h_in[i] = 1;
    }

    int groundTruth = N * 1;

    hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice);

    // allocate block and thread size
    dim3 grid(gridSize);
    dim3 block(blockSize); // thrad size

    // record GPU execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // allocate 1 block, 1 thread
    reduce_v6<blockSize><<<grid, block>>>(d_in, part_out, N);
    reduce_v6<blockSize><<<1, block>>>(d_in, d_out, gridSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(h_out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);
    printf("allocated %d blocks, data counts are %d\n", gridSize, N);

    bool is_right = checkResult(h_out, groundTruth, 1);
    if(is_right)
    {
        printf("the ans is right\n");
    }
    else
    {
        printf("the ans is wrong\n");
        // for(int i = 0; i < gridSize; i++)
        // {
        //     printf("res per block: %1f\n", h_out[i]);
        // }
        printf("groundTruth is %f \n", groundTruth);
    }

    printf("reduce_baseline latency = %f ms\n", milliseconds);

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;

}